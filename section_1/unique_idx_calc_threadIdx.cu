#include "hip/hip_runtime.h"


#include <stdio.h>

__global__ void unique_idx_calc_threadIdx(int *input)
{
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    printf("Idx : %d, value : %d\n", gid, input[gid]);
}

int main()
{
    int h_data[] = {
        23,
        9,
        4,
        53,
        65,
        12,
        1,
        33,
        87,
        45,
        23,
        12,
        342,
        56,
        44,
        99,
    };

    for (int i = 0; i < sizeof(h_data) / sizeof(int); i++)
    {
        printf("%d ", h_data[i]);
    }
    printf("\n \n");

    int *d_data;
    hipMalloc((void **)&d_data, sizeof(h_data));
    hipMemcpy(d_data, h_data, sizeof(h_data), hipMemcpyHostToDevice);

    dim3 block(4);
    dim3 grid(4);
    unique_idx_calc_threadIdx<<<grid, block>>>(d_data);

    hipDeviceSynchronize();
    hipDeviceReset();
    return 0;
}