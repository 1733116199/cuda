#include "hip/hip_runtime.h"


#include <stdio.h>

__global__ void printThreadIds()
{
    printf("ThreadIdx.x : %d, ThreadIdx.y : %d, ThreadIdx.z : %d \n", threadIdx.x, threadIdx.y, threadIdx.z);
}

int main()
{
    int nx = 16, ny = 16;

    dim3 block(8, 8);
    dim3 grid(nx / block.x, ny / block.y);
    printThreadIds<<<grid, block>>>();
    hipDeviceSynchronize();
    hipDeviceReset();
    return 0;
}