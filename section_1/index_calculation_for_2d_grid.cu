#include "hip/hip_runtime.h"


#include <stdio.h>

__global__ void index_calculation_for_2d_grid(int *input)
{
    int gid = (blockIdx.x * gridDim.x + blockIdx.y) * blockDim.x + threadIdx.x;
    printf("blockIdx.x : %d, blockIdx.y: %d, Idx : %d, value : %d\n", blockIdx.x, blockIdx.y, gid, input[gid]);
}

int main()
{
    int h_data[] = {
        23,
        9,
        4,
        53,
        65,
        12,
        1,
        33,
        87,
        45,
        23,
        12,
        342,
        56,
        44,
        99,
    };

    for (int i = 0; i < sizeof(h_data) / sizeof(int); i++)
    {
        printf("%d ", h_data[i]);
    }
    printf("\n \n");

    int *d_data;
    hipMalloc((void **)&d_data, sizeof(h_data));
    hipMemcpy(d_data, h_data, sizeof(h_data), hipMemcpyHostToDevice);

    dim3 block(4);
    dim3 grid(2, 2);
    index_calculation_for_2d_grid<<<grid, block>>>(d_data);

    hipDeviceSynchronize();
    hipDeviceReset();
    return 0;
}