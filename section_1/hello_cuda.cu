#include "hip/hip_runtime.h"


#include <stdio.h>

__global__ void hello_cuda()
{
    printf("Hello CUDA world! \n");
}

int main()
{
    dim3 block(4);
    dim3 grid(8);
    hello_cuda<<<grid, block>>>();
    hipDeviceSynchronize();
    hipDeviceReset();
    return 0;
}